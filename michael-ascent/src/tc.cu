#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>

// https://www.wikiwand.com/en/Floyd%E2%80%93Warshall_algorithm
extern "C" __global__ void tc_kernel(bool *graph, int n) {
    int k = blockIdx.x;
    int i = threadIdx.y;
    int j = threadIdx.x;

    if (i < n && j < n) {
        __syncthreads(); // gotta sync before readin

        if (graph[i * n + k] && graph[k * n + j]) {
            graph[i * n + j] = true;
        }

        __syncthreads(); // gotta sync after updates
    }
}